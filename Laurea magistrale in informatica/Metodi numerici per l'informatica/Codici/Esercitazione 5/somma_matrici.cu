#include <assert.h>
#include <stdio.h>
#include<hip/hip_runtime.h>
#include <time.h>

void sommaCPU(double *A, double *B, double *C, int n);
__global__ void sommaGPU(double *A, double *B, double *C, int n);

int main(void)
{
 double *A_h, *B_h,*C_h,*C2_h; // host data
 double *A_d, *B_d,*C_d; // device data
 int N, nBytes;
dim3 gridDim, blockDim;


printf("***\t  somma di due matrici \t***\n");
printf("Inserire il numero di righe e colonne : \n");
scanf("%d", &N);

 nBytes = N*N*sizeof(double);
 A_h = (double *)malloc(nBytes);
 B_h = (double *)malloc(nBytes);
 C_h = (double *)malloc(nBytes);
 C2_h = (double *)malloc(nBytes);
 hipMalloc((void **) &A_d, nBytes);
 hipMalloc((void **) &B_d, nBytes);
 hipMalloc((void **) &C_d, nBytes);
 // inizializzo i dati
 /*Inizializza la generazione random dei vettori utilizzando l'ora attuale del sistema*/
srand((unsigned int) time(0));


    if(N<=100) printf("A = ");
        for (int i = 0; i < N; i++) {
            for (int j = 0; j < N; j++) {
                if (j == 0) {
                    A_h[i * N + j] = 1.0 / (i + 1) - 1;

                } else {
                    A_h[i * N + j] = 1.0 / (i + 1) - pow(1.0 / 2.0, j);

                }

                if(N<=100)
                printf("   %f  ", A_h[i*N+j] );

            }
            if(N<=100)
            printf("\n\n   ");
        }
        if(N<=100)
        printf("\n\n");

        if(N<=100) printf("B = ");
            for (int i = 0; i < N; i++) {
                for (int j = 0; j < N; j++) {
                    if (j == 0) {
                        B_h[i * N + j] = 2.0 / (i + 1) - 1;

                    } else {
                        B_h[i * N + j] = 2.0 / (i + 1) - pow(1.0 / 2.0, j);

                    }

                    if(N<=100)
                    printf("   %f  ", B_h[i*N+j] );

                }
                if(N<=100)
                printf("\n\n   ");
            }
            if(N<=100)
            printf("\n\n");




 hipMemcpy(A_d, A_h, nBytes, hipMemcpyHostToDevice);
 hipMemcpy(B_d, B_h, nBytes, hipMemcpyHostToDevice);


 //azzeriamo il contenuto del vettore c
memset(C_h, 0, nBytes);
hipMemset(C_d, 0, nBytes);

//configurazione del kernel
blockDim.x=32;
blockDim.y=32;
gridDim.x=N/blockDim.x+((N%blockDim.x)==0?0:1);
gridDim.y=N/blockDim.y+((N%blockDim.y)==0?0:1);


hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);

hipEventRecord(start);

//invocazione del kernel
sommaGPU<<<gridDim, blockDim>>>(A_d, B_d, C_d, N);
hipEventRecord(stop);
hipEventSynchronize(stop); // assicura che tutti siano arrivati all'evento stop prima di registrare il tempo
float elapsed;
// tempo tra i due eventi in millisecondi
hipEventElapsedTime(&elapsed, start, stop);
hipEventDestroy(start);
hipEventDestroy(stop);
printf("tempo GPU=%f\n", elapsed);

 hipMemcpy(C_h, C_d, nBytes, hipMemcpyDeviceToHost);

 // calcolo su CPU
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start);
 // calcolo somma seriale
sommaCPU(A_h, B_h, C2_h, N);

hipEventRecord(stop);
hipEventSynchronize(stop); // assicura che tutti siano arrivati all'evento stop prima di registrare il tempo
hipEventElapsedTime(&elapsed, start, stop);
hipEventDestroy(start);
hipEventDestroy(stop);
printf("tempo CPU=%f\n", elapsed);




 for (int i=0; i< N*N; i++) assert( C_h[i] == C2_h[i] );


 if(N<=100) printf("C_h= ");
     for (int i = 0; i < N; i++) {
         for (int j = 0; j < N; j++) {
             if(N<=100)
             printf("   %f  ", C_h[i*N+j] );
         }
         if(N<=100)
         printf("\n\n   ");
     }
     if(N<=100)
     printf("\n\n");


 free(A_h); free(B_h); free(C_h); free(C2_h);
 hipFree(A_d); hipFree(B_d); hipFree(C_d);
 return 0;
}

//Seriale
void sommaCPU(double *A, double *B, double *C, int n)
{
int i;
for(i=0;i<n*n;i++)
	C[i]=A[i]+B[i];
}

//Parallelo
__global__ void sommaGPU
(double *A, double *B, double *C, int n)
{
int i=threadIdx.x + blockIdx.x*blockDim.x;
int j= blockIdx.y* blockDim.y+ threadIdx.y;
int index= j*gridDim.x* blockDim.x+i;
if(index < n*n)
	C[index] = A[index]+B[index];
}
