#include <assert.h>
#include <stdio.h>
#include<hip/hip_runtime.h>
#include <time.h>

void sommaCPU(double *A, double *B, double *C, int r, int c);
__global__ void sommaGPU(double *A, double *B, double *C, int r,int c);

int main(void)
{
 double *A_h, *B_h,*C_h,*C2_h; // host data
 double *A_d, *B_d,*C_d; // device data
 int righe,colonne, nBytes;
dim3 gridDim, blockDim;


printf("***\t somma di due matrici \t***\n");
printf("Inserire il numero di righe : \n");
scanf("%d", &righe);
printf("Inserire il numero di colonne : \n");
scanf("%d", &colonne);

 nBytes = righe*colonne*sizeof(double);
 A_h = (double *)malloc(nBytes);
 B_h = (double *)malloc(nBytes);
 C_h = (double *)malloc(nBytes);
 C2_h = (double *)malloc(nBytes);
 hipMalloc((void **) &A_d, nBytes);
 hipMalloc((void **) &B_d, nBytes);
 hipMalloc((void **) &C_d, nBytes);
 // inizializzo i dati
 /*Inizializza la generazione random dei vettori utilizzando l'ora attuale del sistema*/
srand((unsigned int) time(0));


    if(righe<=100) printf("A = ");
        for (int i = 0; i < righe; i++) {
            for (int j = 0; j < colonne; j++) {
                if (j == 0) {
                    A_h[i * colonne + j] = 1.0 / (i + 1) - 1;

                } else {
                    A_h[i * colonne + j] = 1.0 / (i + 1) - pow(1.0 / 2.0, j);

                }

                if(righe<=100)
                printf("   %f  ", A_h[i*colonne+j] );

            }
            if(righe<=100) printf("\n\n   ");
        }
        if(righe<=100) printf("\n\n");

        if(righe<=100) printf("B = ");
            for (int i = 0; i < righe; i++) {
                for (int j = 0; j < colonne; j++) {
                    if (j == 0) {
                        B_h[i * colonne + j] = 2.0 / (i + 1) - 1;

                    } else {
                        B_h[i * colonne + j] = 2.0 / (i + 1) - pow(1.0 / 2.0, j);

                    }

                    if(righe<=100)
                    printf("   %f  ", B_h[i*colonne+j] );

                }
                if(righe<=100) printf("\n\n   ");
            }
            if(righe<=100) printf("\n\n");




 hipMemcpy(A_d, A_h, nBytes, hipMemcpyHostToDevice);
 hipMemcpy(B_d, B_h, nBytes, hipMemcpyHostToDevice);


 //azzeriamo il contenuto del vettore c
memset(C_h, 0, nBytes);
hipMemset(C_d, 0, nBytes);

//configurazione del kernel
blockDim.x=32;
blockDim.y=8;
gridDim.x=righe/blockDim.x+((righe%blockDim.x)==0?0:1);
gridDim.y=colonne/blockDim.y+((colonne%blockDim.y)==0?0:1);


hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);

hipEventRecord(start);

//invocazione del kernel
sommaGPU<<<gridDim, blockDim>>>(A_d, B_d, C_d, righe,colonne);
hipEventRecord(stop);
hipEventSynchronize(stop); // assicura che tutti siano arrivati all'evento stop prima di registrare il tempo
float elapsed;
// tempo tra i due eventi in millisecondi
hipEventElapsedTime(&elapsed, start, stop);
hipEventDestroy(start);
hipEventDestroy(stop);
printf("tempo GPU=%f\n", elapsed);

 hipMemcpy(C_h, C_d, nBytes, hipMemcpyDeviceToHost);

 // calcolo su CPU
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start);
 // calcolo somma seriale
sommaCPU(A_h, B_h, C2_h, righe, colonne);

hipEventRecord(stop);
hipEventSynchronize(stop); // assicura che tutti siano arrivati all'evento stop prima di registrare il tempo
hipEventElapsedTime(&elapsed, start, stop);
hipEventDestroy(start);
hipEventDestroy(stop);
printf("tempo CPU=%f\n", elapsed);




 for (int i=0; i< righe*colonne; i++) assert( C_h[i] == C2_h[i] );


 if(righe<=100) printf("C_h= ");
     for (int i = 0; i < righe; i++) {
         for (int j = 0; j < colonne; j++) {
             if(righe<=100)
             printf("   %f  ", C_h[i*colonne+j] );
         }
         if(righe<=100) printf("\n\n   ");
     }
     if(righe<=100) printf("\n\n");


 free(A_h); free(B_h); free(C_h); free(C2_h);
 hipFree(A_d); hipFree(B_d); hipFree(C_d);
 return 0;
}

//Seriale
void sommaCPU(double *A, double *B, double *C, int r, int c)
{
int i;
for(i=0;i<r*c;i++)
	C[i]=A[i]+B[i];
}

//Parallelo
__global__ void sommaGPU
(double *A, double *B, double *C, int r, int c)
{
int i=threadIdx.x + blockIdx.x*blockDim.x;
int j= blockIdx.y* blockDim.y+ threadIdx.y;
int index= j*gridDim.x* blockDim.x+i;
if(index < r*c)
	C[index] = A[index]+B[index];
}
