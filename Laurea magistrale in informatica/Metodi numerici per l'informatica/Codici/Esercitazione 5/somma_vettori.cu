#include <assert.h>
#include <stdio.h>
#include<hip/hip_runtime.h>
#include <time.h>

void sommaCPU(float *a, float *b, float *c, int n);
__global__ void sommaGPU(float* a, float* b, float* c, int n);

int main(void)
{
 float *a_h, *b_h, *c_h, *c_h2; // host data
 float *a_d, *b_d, *c_d; // device data
 int N , nBytes, i ;
dim3 gridDim, blockDim;


printf("***\t somma COMPONENTE PER COMPONENTE DI DUE ARRAY \t***\n");
printf("Inserire il numero di elementi da sommare : \n");
scanf("%d", &N);

 nBytes = N*sizeof(float);
 a_h = (float *)malloc(nBytes);
 b_h = (float *)malloc(nBytes);
 c_h = (float *)malloc(nBytes);
 c_h2 = (float *)malloc(nBytes);
 hipMalloc((void **) &a_d, nBytes);
 hipMalloc((void **) &b_d, nBytes);
 hipMalloc((void **) &c_d, nBytes);
 // inizializzo i dati
 /*Inizializza la generazione random dei vettori utilizzando l'ora attuale del sistema*/
srand((unsigned int) time(0));

 for (i=0; i<N; i++) {
	a_h[i] = rand()%5-2;
	b_h[i] = rand()%5-2;;
 }

 hipMemcpy(a_d, a_h, nBytes, hipMemcpyHostToDevice);
 hipMemcpy(b_d, b_h, nBytes, hipMemcpyHostToDevice);


 //azzeriamo il contenuto del vettore c
memset(c_h, 0, nBytes);
hipMemset(c_d, 0, nBytes);

//configurazione del kernel
blockDim.x=128;
gridDim.x=N/blockDim.x+((N%blockDim.x)==0?0:1);


hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);

hipEventRecord(start);

//invocazione del kernel
sommaGPU<<<gridDim, blockDim>>>(a_d, b_d, c_d, N);
hipEventRecord(stop);
hipEventSynchronize(stop); // assicura che tutti siano arrivati all'evento stop prima di registrare il tempo
float elapsed;
// tempo tra i due eventi in millisecondi
hipEventElapsedTime(&elapsed, start, stop);
hipEventDestroy(start);
hipEventDestroy(stop);
printf("tempo GPU=%f\n", elapsed);

 hipMemcpy(c_h, c_d, nBytes, hipMemcpyDeviceToHost);

 // calcolo su CPU
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start);
 // calcolo somma seriale
sommaCPU(a_h, b_h, c_h2, N);

hipEventRecord(stop);
hipEventSynchronize(stop); // assicura che tutti siano arrivati all'evento stop prima di registrare il tempo
hipEventElapsedTime(&elapsed, start, stop);
hipEventDestroy(start);
hipEventDestroy(stop);
printf("tempo CPU=%f\n", elapsed);




 for (i=0; i< N; i++) assert( c_h[i] == c_h2[i] );

 if (N<20){
 	for(i=0;i<N;i++)
		printf("a_h[%d]=%6.2f ",i, a_h[i]);
	printf("\n");
	for(i=0;i<N;i++)
		printf("b_h[%d]=%6.2f ",i, b_h[i]);
	printf("\n");
	for(i=0;i<N;i++)
		printf("c_h[%d]=%6.2f ",i, c_h[i]);
	printf("\n");
}
 free(a_h); free(b_h); free(c_h); free(c_h2);
 hipFree(a_d); hipFree(b_d); hipFree(c_d);
 return 0;
}

//Seriale
void sommaCPU(float *a, float *b, float *c, int n)
{
int i;
for(i=0;i<n;i++)
	c[i]=a[i]+b[i];
}

//Parallelo
__global__ void sommaGPU
(float* a, float * b, float* c, int n)
{
int index=threadIdx.x + blockIdx.x*blockDim.x;
if(index < n)
	c[index] = a[index]+b[index];
}
