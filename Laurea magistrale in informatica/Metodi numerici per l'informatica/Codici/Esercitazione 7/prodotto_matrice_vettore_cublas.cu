#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>


// alfa = 1, beta e y = 0
void productMatrixVectorCPU(float *a, float *b, float *v, int N);
void generateMatrix(float *h_A, int  n);

int main (void){
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    float* h_A = 0;       // Host matrice A
    float* d_A;           // Device matrice A
    float* h_b = 0;       // Host array b
    float* d_b;           // Device array b
    float* d_v;           // Device risultato prodotto v
    float* h_v2= 0;       // Risultato finale del prodotto matrice vettore portato da device a host
    float* h_v;           // Risultato cpu
    float alfa = 1, beta = 0;
	int N,i;

    printf("***\t prodotto matrice vettore con cublas \t***\n");
    printf("Inserire numero di righe e colonne : \n");
    scanf("%d", &N);


    h_A = (float *)malloc (N*N* sizeof (*h_A));      // Alloco h_A e lo inizializzo
    if (!h_A) {
        printf ("host memory allocation failed");
        return EXIT_FAILURE;
    }

    generateMatrix(h_A, N);

    h_b = (float *)malloc (N * sizeof (*h_b));  // Alloco h_b e lo inizializzo
    if (!h_b) {
        printf ("host memory allocation failed");
        return EXIT_FAILURE;
    }

    for (i=0; i<N; i++) h_b[i] = rand()%5-2;


    cudaStat = hipMalloc ((void**)&d_A, N*N*sizeof(*h_A));       // Alloco d_A
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed");
        return EXIT_FAILURE;
    }

    cudaStat = hipMalloc ((void**)&d_b, N*sizeof(*h_b));       // Alloco d_b
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed");
        return EXIT_FAILURE;
    }

    stat = hipblasCreate(&handle);               // Creo l'handle per cublas
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }

    stat = hipblasSetMatrix(N,N,sizeof(float),h_A,N,d_A,N);    // Setto h_A su d_A
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed");
        hipFree (d_A);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    stat = hipblasSetVector(N,sizeof(float),h_b,1,d_b,1);    // Setto h_b su d_b
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed");
        hipFree (d_b);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    h_v2 = (float *)malloc(N*sizeof(float));
    memset(h_v2, 0, N*sizeof(float));
    
    cudaStat = hipMalloc ((void**)&d_v, N*sizeof(*h_v2));       // Alloco d_v
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed");
        return EXIT_FAILURE;
    }
    
    stat = hipblasSetVector(N,sizeof(float),h_v2,1,d_v,1);    // Setto h_b su d_b
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed");
        hipFree (d_v);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }

    // parte calcolo GPU
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    stat = hipblasSgemv(handle,HIPBLAS_OP_T,N,N,&alfa,d_A,N,d_b,1,&beta,d_v,1);

    hipEventRecord(stop);
    hipEventSynchronize(stop); // assicura che tutti siano arrivati all'evento stop prima di registrare il tempo
    float elapsed;
    // tempo tra i due eventi in millisecondi
    hipEventElapsedTime(&elapsed, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("tempo GPU=%f\n", elapsed);

    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed hipblasSdot");
        hipFree (d_A);
        hipFree (d_b);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }


    // calcolo su CPU
    h_v = (float *)malloc(N*sizeof(float));
    memset(h_v, 0, N*sizeof(float));

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

     // calcolo somma seriale
    productMatrixVectorCPU(h_A, h_b, h_v, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop); // assicura che tutti siano arrivati all'evento stop prima di registrare il tempo
    hipEventElapsedTime(&elapsed, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("tempo CPU=%f\n", elapsed);


    stat = hipblasGetVector(N,sizeof(float),d_v,1,h_v2,1); 
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed hipblasGetVector");
        hipFree (d_A);
        hipFree (d_b);
        hipFree (d_v);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    
    hipFree (d_A);     // Dealloco d_a
    hipFree (d_b);     // Dealloco d_b
    hipFree (d_v);     // Dealloco d_v

    hipblasDestroy(handle);  // Distruggo l'handle

    free(h_A);      // Dealloco h_a
    free(h_b);      // Dealloco h_b
    free(h_v);      // Dealloco h_v
    free(h_v2);      // Dealloco h_v
    return EXIT_SUCCESS;
}

void generateMatrix(float *h_A, int N){
  if(N<=20) printf("A = ");
    for (int i = 0; i < N; i++) {
      for (int j = 0; j < N; j++) {
        if (j == 0) h_A[i * N + j] = 1.0 / (i + 1) - 1;
        else h_A[i * N + j] = 1.0 / (i + 1) - pow(1.0 / 2.0, j);
        if(N<=20) printf("   %f  ", h_A[i*N+j] );
      }
      if(N<=20) printf("\n\n   ");
    }
    if(N<=20) printf("\n\n");
}

//Seriale
void productMatrixVectorCPU(float *a, float *b, float *v, int n){
    int i,j;
    for(i = 0; i < n; i++)
      for(j = 0; j < n; j++)
        v[i] += a[i * n + j] * b[j];
}
