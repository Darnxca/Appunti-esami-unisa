#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

void prodottoScalareCPU(float *a, float *b, float *v, int n);

int main (void){
    hipError_t cudaStat;
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    float* h_a = 0;     // Host array a
    float* d_a;         // Device array a
    float* h_b = 0;     // Host array b
    float *d_b;         // Device array b
    float somma_gpu = 0;   // Risultato finale
    float* h_v;         // Risultato cpu
    float somma_cpu=0;
	int M,i;
	
    printf("***\t prodotto scalare con cublas \t***\n");
    printf("Inserire numero di elementi : \n");
    scanf("%d", &M);
    
    h_a = (float *)malloc (M * sizeof (*h_a));      // Alloco h_a e lo inizializzo
    if (!h_a) {
        printf ("host memory allocation failed");
        return EXIT_FAILURE;
    }
    
    
    h_b = (float *)malloc (M * sizeof (*h_b));  // Alloco h_b e lo inizializzo
    if (!h_b) {
        printf ("host memory allocation failed");
        return EXIT_FAILURE;
    }
    
    for (i=0; i<M; i++) {
        h_a[i] = rand()%5-2;
    	h_b[i] = rand()%5-2;
    }
    
    cudaStat = hipMalloc ((void**)&d_a, M*sizeof(*h_a));       // Alloco d_a
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed");
        return EXIT_FAILURE;
    }
    
    cudaStat = hipMalloc ((void**)&d_b, M*sizeof(*h_b));       // Alloco d_b
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed");
        return EXIT_FAILURE;
    }
    
    stat = hipblasCreate(&handle);               // Creo l'handle per cublas
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }
    
    stat = hipblasSetVector(M,sizeof(float),h_a,1,d_a,1);    // Setto h_a su d_a
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed");
        hipFree (d_a);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    
    stat = hipblasSetVector(M,sizeof(float),h_b,1,d_b,1);    // Setto h_b su d_b
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed");
        hipFree (d_b);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    
    // parte calcolo GPU
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    
    stat = hipblasSdot(handle,M,d_a,1,d_b,1,&somma_gpu);        // Calcolo il prodotto
    
    hipEventRecord(stop);
    hipEventSynchronize(stop); // assicura che tutti siano arrivati all'evento stop prima di registrare il tempo
    float elapsed;
    // tempo tra i due eventi in millisecondi
    hipEventElapsedTime(&elapsed, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("tempo GPU=%f\n", elapsed);
    
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed hipblasSdot");
        hipFree (d_a);
        hipFree (d_b);
        hipblasDestroy(handle);
        return EXIT_FAILURE;
    }
    
    
    // calcolo su CPU
    h_v = (float *)malloc(M*sizeof(float));
    
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
     // calcolo somma seriale
    prodottoScalareCPU(h_a, h_b, h_v, M);
    
    for(i=0;i<M;i++){
    	somma_cpu=somma_cpu+h_v[i];         
    }
    
    hipEventRecord(stop);
    hipEventSynchronize(stop); // assicura che tutti siano arrivati all'evento stop prima di registrare il tempo
    hipEventElapsedTime(&elapsed, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("tempo CPU=%f\n", elapsed);
    
    
    assert(somma_cpu == somma_gpu);
    
    printf("Risultato del prodotto --> %f",somma_gpu);
    
    hipFree (d_a);     // Dealloco d_a
    hipFree (d_b);     // Dealloco d_b
    
    hipblasDestroy(handle);  // Distruggo l'handle
    
    free(h_a);      // Dealloco h_a
    free(h_b);      // Dealloco h_b    
    free(h_v);      // Dealloco h_v  
    return EXIT_SUCCESS;
}

//Seriale
void prodottoScalareCPU(float *a, float *b, float *v, int n){
    int i;
    for(i=0;i<n;i++)
    	v[i]=a[i]*b[i];
}