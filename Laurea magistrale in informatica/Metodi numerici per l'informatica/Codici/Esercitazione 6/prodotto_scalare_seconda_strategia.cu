/*Inserire N come multiplo di 128*/
#include <assert.h>
#include <stdio.h>
#include<hip/hip_runtime.h>
#include <time.h>

void prodottoScalareCPU(float *a, float *b, float *v, int n);
__global__ void prodottoScalareGPU(float* a, float* b, float* c, int n);

int main(void){
    float *a_h, *b_h, *v_h, *v_h2; // host data
    float *a_d, *b_d, *v_d; // device data
    float somma_cpu=0;
    float somma_gpu=0;
    int N , nBytes, i,sharedDim ;
    dim3 gridDim, blockDim;


    printf("***\t prodotto scalare seconda strategia \t***\n");
    printf("Inserire numero di elementi array : \n");
    scanf("%d", &N);

    nBytes = N*sizeof(float);
    a_h = (float *)malloc(nBytes);
    b_h = (float *)malloc(nBytes);
    v_h = (float *)malloc(nBytes);
    v_h2 = (float *)malloc(nBytes);
    hipMalloc((void **) &a_d, nBytes);
    hipMalloc((void **) &b_d, nBytes);
    hipMalloc((void **) &v_d, nBytes);
    // inizializzo i dati
    /*Inizializza la generazione random dei vettori utilizzando l'ora attuale del sistema*/
    srand((unsigned int) time(0));

    for (i=0; i<N; i++) {
        a_h[i] = rand()%5-2;
    	b_h[i] = rand()%5-2;;
    }

    hipMemcpy(a_d, a_h, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(b_d, b_h, nBytes, hipMemcpyHostToDevice);


    //azzeriamo il contenuto del vettore v
    memset(v_h, 0, gridDim.x * sizeof(float));
    hipMemset(v_d, 0, nBytes);

    //configurazione del kernel
    blockDim.x=128;
    gridDim.x=N/blockDim.x+((N%blockDim.x)==0?0:1);
    // dimensione dell' array condiviso fra i thread allocato dinamicamente
    sharedDim = blockDim.x*sizeof(float);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    //invocazione del kernel
    prodottoScalareGPU<<<gridDim, blockDim, sharedDim>>>(a_d, b_d, v_d, N);

    hipMemcpy(v_h, v_d, gridDim.x * sizeof(float), hipMemcpyDeviceToHost);

    for(i=0;i<gridDim.x;i++){
    	somma_gpu=somma_gpu+v_h[i];
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop); // assicura che tutti siano arrivati all'evento stop prima di registrare il tempo
    float elapsed;
    // tempo tra i due eventi in millisecondi
    hipEventElapsedTime(&elapsed, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("tempo GPU=%f\n", elapsed);

    //cudaMemcpy(v_h, v_d, nBytes, cudaMemcpyDeviceToHost);

     // calcolo su CPU
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
     // calcolo somma seriale
    prodottoScalareCPU(a_h, b_h, v_h2, N);

    for(i=0;i<N;i++){
    	somma_cpu=somma_cpu+v_h2[i];
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop); // assicura che tutti siano arrivati all'evento stop prima di registrare il tempo
    hipEventElapsedTime(&elapsed, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("tempo CPU=%f\n", elapsed);


    assert( somma_cpu == somma_gpu);

    if (N<20){
        for(i=0;i<N;i++)
    	    printf("a_h[%d]=%6.2f ",i, a_h[i]);
    	printf("\n");
    	for(i=0;i<N;i++)
    		printf("b_h[%d]=%6.2f ",i, b_h[i]);
    	printf("\n");
    	for(i=0;i<N;i++)
    		printf("v_h[%d]=%6.2f ",i, v_h[i]);
    	printf("\n");
    }

    printf("Il prodotto scalare sulla gpu fra a e b = %f",somma_gpu);
    printf("\n");

    printf("Il prodotto scalare sulla cpu fra a e b = %f",somma_cpu);
    printf("\n");

    free(a_h); free(b_h); free(v_h); free(v_h2);
    hipFree(a_d); hipFree(b_d); hipFree(v_d);
    return 0;
}

//Seriale
void prodottoScalareCPU(float *a, float *b, float *v, int n){
    int i;
    for(i=0;i<n;i++)
    	v[i]=a[i]*b[i];
}

//Parallelo
__global__ void prodottoScalareGPU(float* a, float * b, float* c, int n){
    int p, Dist,passi = 0;

    extern __shared__ float v[];
    int index=threadIdx.x + blockIdx.x*blockDim.x;
    int id = threadIdx.x;

    if(index < n)
        v[id] = a[index]*b[index];
    __syncthreads();

    // somma in parallelo
    p = blockDim.x;
    while(p!=1){/*shifta di un bit a destra*/
		p=p>>1; //divido
		passi++;
	}

	for(int i=0;i<passi;i++){
	    Dist = 1<<i; // moltiplico 2^i

	    if(id %(Dist * 2) == 0){
	        v[id] = v[id] + v[id + Dist];
	    }

	    __syncthreads();
	}

	if(id == 0) c[blockIdx.x] = v[0];
}
